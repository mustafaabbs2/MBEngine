#include "hip/hip_runtime.h"
#include <array>
#include <hip/hip_runtime.h>
#include <iostream>
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_stubs.h>
#include <sstream>
#include <stdexcept>

#include "optixTriangle.h"

#define CUDA_CHECK(call) cudaCheck(call, #call, __FILE__, __LINE__)
#define OPTIX_CHECK(call) optixCheck(call, #call, __FILE__, __LINE__)

inline void cudaCheck(hipError_t error, const char* call, const char* file, unsigned int line)
{
	if(error != hipSuccess)
	{
		std::stringstream ss;
		ss << "CUDA call (" << call << " ) failed with error: '" << hipGetErrorString(error)
		   << "' (" << file << ":" << line << ")\n";
		throw std::runtime_error(ss.str().c_str());
	}
}

inline void optixCheck(OptixResult res, const char* call, const char* file, unsigned int line)
{
	if(res != OPTIX_SUCCESS)
	{
		std::stringstream ss;
		ss << "Optix call '" << call << "' failed: " << file << ':' << line << ")\n";
		throw std::runtime_error(ss.str().c_str());
	}
}

template <typename T>
struct SbtRecord
{
	__align__(OPTIX_SBT_RECORD_ALIGNMENT) char header[OPTIX_SBT_RECORD_HEADER_SIZE];
	T data;
};

typedef SbtRecord<RayGenData> RayGenSbtRecord;
typedef SbtRecord<MissData> MissSbtRecord;
typedef SbtRecord<HitGroupData> HitGroupSbtRecord;

int main()
{

	int width = 1024;
	int height = 768;

	// Initialize CUDA
	hipCtx_t cuCtx = 0; // zero means take the current context

	// Applications wishing to leverage multiple GPUs in a system may create multiple device contexts and use them to invoke independent launches per device.

	CUDA_CHECK(hipFree(0)); // Initialize CUDA

	//A good practice is to zero-initialize all OptiX input structs to mark all fields as default, then to selectively override the fields to be used.

	// Create an OptiX device context
	OptixDeviceContext context = nullptr;
	OptixDeviceContextOptions options = {};
	options.logCallbackFunction = nullptr;
	options.logCallbackLevel = 4;

	hipError_t cuRes = hipCtxGetCurrent(&cuCtx);
	if(cuRes != hipSuccess)
	{
		std::cerr << "Error getting current CUDA context" << std::endl;
		return 1;
	}

	OPTIX_CHECK(optixInit());

	OptixResult optixRes = optixDeviceContextCreate(cuCtx, &options, &context);
	if(optixRes != OPTIX_SUCCESS)
	{
		std::cerr << "Error creating OptiX device context" << std::endl;
		return 1;
	}

	std::cout << "Hello, OptiX!" << std::endl;

	//now do your thing:

	//Define geometry and create GAS
	OptixTraversableHandle gas_handle;
	hipDeviceptr_t d_gas_output_buffer;
	{
		// Use default options for simplicity.  In a real use case we would want to
		// enable compaction, etc
		OptixAccelBuildOptions accel_options = {};
		accel_options.buildFlags = OPTIX_BUILD_FLAG_NONE;
		accel_options.operation = OPTIX_BUILD_OPERATION_BUILD;

		// Triangle build input: simple list of three vertices
		const std::array<float3, 3> vertices = {
			{{-0.5f, -0.5f, 0.0f}, {0.5f, -0.5f, 0.0f}, {0.0f, 0.5f, 0.0f}}};

		const size_t vertices_size = sizeof(float3) * vertices.size();
		hipDeviceptr_t d_vertices = 0;
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_vertices), vertices_size));
		CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(d_vertices),
							  vertices.data(),
							  vertices_size,
							  hipMemcpyHostToDevice));

		const uint32_t triangle_input_flags[1] = {OPTIX_GEOMETRY_FLAG_NONE};
		OptixBuildInput triangle_input = {};
		triangle_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
		triangle_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
		triangle_input.triangleArray.numVertices = static_cast<uint32_t>(vertices.size());
		triangle_input.triangleArray.vertexBuffers = &d_vertices;
		triangle_input.triangleArray.flags = triangle_input_flags;
		triangle_input.triangleArray.numSbtRecords = 1;

		OptixAccelBufferSizes gas_buffer_sizes;
		OPTIX_CHECK(optixAccelComputeMemoryUsage(context,
												 &accel_options,
												 &triangle_input,
												 1, // Number of build inputs
												 &gas_buffer_sizes));
		hipDeviceptr_t d_temp_buffer_gas;
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_temp_buffer_gas),
							  gas_buffer_sizes.tempSizeInBytes));
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_gas_output_buffer),
							  gas_buffer_sizes.outputSizeInBytes));

		OPTIX_CHECK(optixAccelBuild(context,
									0, // CUDA stream
									&accel_options,
									&triangle_input,
									1, // num build inputs
									d_temp_buffer_gas,
									gas_buffer_sizes.tempSizeInBytes,
									d_gas_output_buffer,
									gas_buffer_sizes.outputSizeInBytes,
									&gas_handle,
									nullptr, // emitted property list
									0 // num emitted properties
									));

		// We can now free the scratch space buffer used during build and the vertex
		// inputs, since they are not needed by our trivial shading method
		CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_temp_buffer_gas)));
		CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_vertices)));
	}

	//
	// Create module
	//
	OptixModule module = nullptr;
	OptixPipelineCompileOptions pipeline_compile_options = {};
	{
		OptixModuleCompileOptions module_compile_options = {};
		module_compile_options.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
		module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
		pipeline_compile_options.usesMotionBlur = false;
		pipeline_compile_options.traversableGraphFlags =
			OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
		pipeline_compile_options.numPayloadValues = 3;
		pipeline_compile_options.numAttributeValues = 3;
		pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
		pipeline_compile_options.pipelineLaunchParamsVariableName = "params";
		pipeline_compile_options.usesPrimitiveTypeFlags = OPTIX_PRIMITIVE_TYPE_FLAGS_TRIANGLE;

		//get input data
		// size_t inputSize = 0;
		// const char* input =
		// 	sutil::getInputData(OPTIX_SAMPLE_NAME, OPTIX_SAMPLE_DIR, "optixTriangle.cu", inputSize);

		// OPTIX_CHECK_LOG(optixModuleCreate(context,
		// 								  &module_compile_options,
		// 								  &pipeline_compile_options,
		// 								  input,
		// 								  inputSize,
		// 								  LOG,
		// 								  &LOG_SIZE,
		// 								  &module));
	}

	//
	// Create program groups
	//
	OptixProgramGroup raygen_prog_group = nullptr;
	OptixProgramGroup miss_prog_group = nullptr;
	OptixProgramGroup hitgroup_prog_group = nullptr;
	{
		OptixProgramGroupOptions program_group_options = {}; // Initialize to zeros

		OptixProgramGroupDesc raygen_prog_group_desc = {}; //
		raygen_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
		raygen_prog_group_desc.raygen.module = module;
		raygen_prog_group_desc.raygen.entryFunctionName = "__raygen__rg";

		char log[2048];
		size_t sizeof_log = sizeof(log);

		optixProgramGroupCreate(context,
								&raygen_prog_group_desc,
								1, // num program groups
								&program_group_options,
								log,
								&sizeof_log,
								&raygen_prog_group);

		OptixProgramGroupDesc miss_prog_group_desc = {};
		miss_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
		miss_prog_group_desc.miss.module = module;
		miss_prog_group_desc.miss.entryFunctionName = "__miss__ms";
		optixProgramGroupCreate(context,
								&miss_prog_group_desc,
								1, // num program groups
								&program_group_options,
								log,
								&sizeof_log,
								&miss_prog_group);

		OptixProgramGroupDesc hitgroup_prog_group_desc = {};
		hitgroup_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
		hitgroup_prog_group_desc.hitgroup.moduleCH = module;
		hitgroup_prog_group_desc.hitgroup.entryFunctionNameCH = "__closesthit__ch";
		optixProgramGroupCreate(context,
								&hitgroup_prog_group_desc,
								1, // num program groups
								&program_group_options,
								log,
								&sizeof_log,
								&hitgroup_prog_group);
	}

	//
	// Link pipeline
	//
	OptixPipeline pipeline = nullptr;
	{
		const uint32_t max_trace_depth = 1;
		OptixProgramGroup program_groups[] = {
			raygen_prog_group, miss_prog_group, hitgroup_prog_group};

		char log[2048];
		size_t sizeof_log = sizeof(log);

		OptixPipelineLinkOptions pipeline_link_options = {};
		pipeline_link_options.maxTraceDepth = max_trace_depth;
		optixPipelineCreate(context,
							&pipeline_compile_options,
							&pipeline_link_options,
							program_groups,
							sizeof(program_groups) / sizeof(program_groups[0]),
							log,
							&sizeof_log,
							&pipeline);
	}

	//
	// Set up shader binding table
	//
	OptixShaderBindingTable sbt = {};
	{
		hipDeviceptr_t raygen_record;
		const size_t raygen_record_size = sizeof(RayGenSbtRecord);
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&raygen_record), raygen_record_size));
		RayGenSbtRecord rg_sbt;
		OPTIX_CHECK(optixSbtRecordPackHeader(raygen_prog_group, &rg_sbt));
		CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(raygen_record),
							  &rg_sbt,
							  raygen_record_size,
							  hipMemcpyHostToDevice));

		hipDeviceptr_t miss_record;
		size_t miss_record_size = sizeof(MissSbtRecord);
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&miss_record), miss_record_size));
		MissSbtRecord ms_sbt;
		ms_sbt.data = {0.3f, 0.1f, 0.2f};
		OPTIX_CHECK(optixSbtRecordPackHeader(miss_prog_group, &ms_sbt));
		CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(miss_record),
							  &ms_sbt,
							  miss_record_size,
							  hipMemcpyHostToDevice));

		hipDeviceptr_t hitgroup_record;
		size_t hitgroup_record_size = sizeof(HitGroupSbtRecord);
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&hitgroup_record), hitgroup_record_size));
		HitGroupSbtRecord hg_sbt;
		OPTIX_CHECK(optixSbtRecordPackHeader(hitgroup_prog_group, &hg_sbt));
		CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(hitgroup_record),
							  &hg_sbt,
							  hitgroup_record_size,
							  hipMemcpyHostToDevice));

		sbt.raygenRecord = raygen_record;
		sbt.missRecordBase = miss_record;
		sbt.missRecordStrideInBytes = sizeof(MissSbtRecord);
		sbt.missRecordCount = 1;
		sbt.hitgroupRecordBase = hitgroup_record;
		sbt.hitgroupRecordStrideInBytes = sizeof(HitGroupSbtRecord);
		sbt.hitgroupRecordCount = 1;
	}

	uchar4* d_output_buffer;
	size_t buffer_size = width * height * sizeof(uchar4);

	// Allocate device memory for output buffer
	CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_output_buffer), buffer_size));

	// launch and get back results

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	Params params;
	params.image = d_output_buffer;
	params.image_width = width;
	params.image_height = height;
	params.handle = gas_handle;

	params.cam_u = make_float3(1.0f, 0.0f, 0.0f); // Example: X axis
	params.cam_v = make_float3(0.0f, 1.0f, 0.0f); // Example: Y axis
	params.cam_w =
		make_float3(0.0f, 0.0f, -1.0f); // Example: -Z axis (assuming camera looks along -Z)
	params.cam_eye = make_float3(0.0f, 0.0f, 1.0f); // Example: Camera position at (0, 0, 1)

	hipDeviceptr_t d_param;
	CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_param), sizeof(Params)));
	CUDA_CHECK(hipMemcpy(
		reinterpret_cast<void*>(d_param), &params, sizeof(params), hipMemcpyHostToDevice));

	OPTIX_CHECK(
		optixLaunch(pipeline, stream, d_param, sizeof(Params), &sbt, width, height, /*depth=*/1));

	CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_param)));

	// Copy results back from device to host if needed
	uchar4* h_output_buffer = new uchar4[width * height];
	CUDA_CHECK(hipMemcpy(h_output_buffer, d_output_buffer, buffer_size, hipMemcpyDeviceToHost));

	// Process or save the output as needed

	// Free device memory for output buffer
	CUDA_CHECK(hipFree(d_output_buffer));

	//display buffer in glfw/image

	//delete
	delete[] h_output_buffer;

	// Destroy CUDA stream
	CUDA_CHECK(hipStreamDestroy(stream));

	optixDeviceContextDestroy(context);

	return 0;
}
