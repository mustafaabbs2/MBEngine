#include "hip/hip_runtime.h"
#include <array>
#include <hip/hip_runtime.h>
#include <iostream>
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_stubs.h>
#include <sstream>
#include <stdexcept>

#define CUDA_CHECK(call) cudaCheck(call, #call, __FILE__, __LINE__)
#define OPTIX_CHECK(call) optixCheck(call, #call, __FILE__, __LINE__)

inline void cudaCheck(hipError_t error, const char* call, const char* file, unsigned int line)
{
	if(error != hipSuccess)
	{
		std::stringstream ss;
		ss << "CUDA call (" << call << " ) failed with error: '" << hipGetErrorString(error)
		   << "' (" << file << ":" << line << ")\n";
		throw std::runtime_error(ss.str().c_str());
	}
}

inline void optixCheck(OptixResult res, const char* call, const char* file, unsigned int line)
{
	if(res != OPTIX_SUCCESS)
	{
		std::stringstream ss;
		ss << "Optix call '" << call << "' failed: " << file << ':' << line << ")\n";
		throw std::runtime_error(ss.str().c_str());
	}
}

int main()
{
	// Initialize CUDA
	hipCtx_t cuCtx = 0; // zero means take the current context

	// Applications wishing to leverage multiple GPUs in a system may create multiple device contexts and use them to invoke independent launches per device.

	CUDA_CHECK(hipFree(0)); // Initialize CUDA

	//A good practice is to zero-initialize all OptiX input structs to mark all fields as default, then to selectively override the fields to be used.

	// Create an OptiX device context
	OptixDeviceContext context = nullptr;
	OptixDeviceContextOptions options = {};
	options.logCallbackFunction = nullptr;
	options.logCallbackLevel = 4;

	hipError_t cuRes = hipCtxGetCurrent(&cuCtx);
	if(cuRes != hipSuccess)
	{
		std::cerr << "Error getting current CUDA context" << std::endl;
		return 1;
	}

	OPTIX_CHECK(optixInit());

	OptixResult optixRes = optixDeviceContextCreate(cuCtx, &options, &context);
	if(optixRes != OPTIX_SUCCESS)
	{
		std::cerr << "Error creating OptiX device context" << std::endl;
		return 1;
	}

	std::cout << "Hello, OptiX!" << std::endl;

	//now do your thing:

	//Define geometry and create GAS
	OptixTraversableHandle gas_handle;
	hipDeviceptr_t d_gas_output_buffer;
	{
		// Use default options for simplicity.  In a real use case we would want to
		// enable compaction, etc
		OptixAccelBuildOptions accel_options = {};
		accel_options.buildFlags = OPTIX_BUILD_FLAG_NONE;
		accel_options.operation = OPTIX_BUILD_OPERATION_BUILD;

		// Triangle build input: simple list of three vertices
		const std::array<float3, 3> vertices = {
			{{-0.5f, -0.5f, 0.0f}, {0.5f, -0.5f, 0.0f}, {0.0f, 0.5f, 0.0f}}};

		const size_t vertices_size = sizeof(float3) * vertices.size();
		hipDeviceptr_t d_vertices = 0;
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_vertices), vertices_size));
		CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(d_vertices),
							  vertices.data(),
							  vertices_size,
							  hipMemcpyHostToDevice));

		const uint32_t triangle_input_flags[1] = {OPTIX_GEOMETRY_FLAG_NONE};
		OptixBuildInput triangle_input = {};
		triangle_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;
		triangle_input.triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
		triangle_input.triangleArray.numVertices = static_cast<uint32_t>(vertices.size());
		triangle_input.triangleArray.vertexBuffers = &d_vertices;
		triangle_input.triangleArray.flags = triangle_input_flags;
		triangle_input.triangleArray.numSbtRecords = 1;

		OptixAccelBufferSizes gas_buffer_sizes;
		OPTIX_CHECK(optixAccelComputeMemoryUsage(context,
												 &accel_options,
												 &triangle_input,
												 1, // Number of build inputs
												 &gas_buffer_sizes));
		hipDeviceptr_t d_temp_buffer_gas;
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_temp_buffer_gas),
							  gas_buffer_sizes.tempSizeInBytes));
		CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_gas_output_buffer),
							  gas_buffer_sizes.outputSizeInBytes));

		OPTIX_CHECK(optixAccelBuild(context,
									0, // CUDA stream
									&accel_options,
									&triangle_input,
									1, // num build inputs
									d_temp_buffer_gas,
									gas_buffer_sizes.tempSizeInBytes,
									d_gas_output_buffer,
									gas_buffer_sizes.outputSizeInBytes,
									&gas_handle,
									nullptr, // emitted property list
									0 // num emitted properties
									));

		// We can now free the scratch space buffer used during build and the vertex
		// inputs, since they are not needed by our trivial shading method
		CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_temp_buffer_gas)));
		CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_vertices)));
	}

	//
	// Create module
	//
	OptixModule module = nullptr;
	OptixPipelineCompileOptions pipeline_compile_options = {};
	{
		OptixModuleCompileOptions module_compile_options = {};
		module_compile_options.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
		module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;
		pipeline_compile_options.usesMotionBlur = false;
		pipeline_compile_options.traversableGraphFlags =
			OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
		pipeline_compile_options.numPayloadValues = 3;
		pipeline_compile_options.numAttributeValues = 3;
		pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
		pipeline_compile_options.pipelineLaunchParamsVariableName = "params";
		pipeline_compile_options.usesPrimitiveTypeFlags = OPTIX_PRIMITIVE_TYPE_FLAGS_TRIANGLE;

		//get input data
		// size_t inputSize = 0;
		// const char* input =
		// 	sutil::getInputData(OPTIX_SAMPLE_NAME, OPTIX_SAMPLE_DIR, "optixTriangle.cu", inputSize);

		// OPTIX_CHECK_LOG(optixModuleCreate(context,
		// 								  &module_compile_options,
		// 								  &pipeline_compile_options,
		// 								  input,
		// 								  inputSize,
		// 								  LOG,
		// 								  &LOG_SIZE,
		// 								  &module));
	}

	//
	// Create program groups
	//
	OptixProgramGroup raygen_prog_group = nullptr;
	OptixProgramGroup miss_prog_group = nullptr;
	OptixProgramGroup hitgroup_prog_group = nullptr;
	{
		OptixProgramGroupOptions program_group_options = {}; // Initialize to zeros

		OptixProgramGroupDesc raygen_prog_group_desc = {}; //
		raygen_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
		raygen_prog_group_desc.raygen.module = module;
		raygen_prog_group_desc.raygen.entryFunctionName = "__raygen__rg";

		char log[2048];
		size_t sizeof_log = sizeof(log);

		optixProgramGroupCreate(context,
								&raygen_prog_group_desc,
								1, // num program groups
								&program_group_options,
								log,
								&sizeof_log,
								&raygen_prog_group);

		OptixProgramGroupDesc miss_prog_group_desc = {};
		miss_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
		miss_prog_group_desc.miss.module = module;
		miss_prog_group_desc.miss.entryFunctionName = "__miss__ms";
		optixProgramGroupCreate(context,
								&miss_prog_group_desc,
								1, // num program groups
								&program_group_options,
								log,
								&sizeof_log,
								&miss_prog_group);

		OptixProgramGroupDesc hitgroup_prog_group_desc = {};
		hitgroup_prog_group_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
		hitgroup_prog_group_desc.hitgroup.moduleCH = module;
		hitgroup_prog_group_desc.hitgroup.entryFunctionNameCH = "__closesthit__ch";
		optixProgramGroupCreate(context,
								&hitgroup_prog_group_desc,
								1, // num program groups
								&program_group_options,
								log,
								&sizeof_log,
								&hitgroup_prog_group);
	}

	optixDeviceContextDestroy(context);

	return 0;
}
