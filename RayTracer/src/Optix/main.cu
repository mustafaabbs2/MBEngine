#include <hip/hip_runtime.h>
#include <iostream>
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_stubs.h>

int main()
{
	// Initialize CUDA
	hipCtx_t cuCtx = 0; // zero means take the current context
	hipFree(0); // Initialize CUDA

	// Create an OptiX device context
	OptixDeviceContext context = nullptr;
	OptixDeviceContextOptions options = {};
	options.logCallbackFunction = nullptr;
	options.logCallbackLevel = 4;

	hipError_t cuRes = hipCtxGetCurrent(&cuCtx);
	if(cuRes != hipSuccess)
	{
		std::cerr << "Error getting current CUDA context" << std::endl;
		return 1;
	}

	OptixResult optixRes = optixDeviceContextCreate(cuCtx, &options, &context);
	if(optixRes != OPTIX_SUCCESS)
	{
		std::cerr << "Error creating OptiX device context" << std::endl;
		return 1;
	}

	std::cout << "Hello, OptiX!" << std::endl;

	optixDeviceContextDestroy(context);

	return 0;
}

