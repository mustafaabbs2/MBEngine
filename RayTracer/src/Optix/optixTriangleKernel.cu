#include "hip/hip_runtime.h"
#include <optix.h>

#include "optixTriangle.h"

extern "C"
{
	__constant__ Params params;
}

static __forceinline__ __device__ void setPayload(float3 p)
{
	optixSetPayload_0(__float_as_uint(p.x));
	optixSetPayload_1(__float_as_uint(p.y));
	optixSetPayload_2(__float_as_uint(p.z));
}

__host__ __device__ float3 make_float3(const float2& a)
{
	return make_float3(a.x, a.y, 0.0f);
}

__host__ __device__ float3 make_float3(const float2& v0, const float v1)
{
	return make_float3(v0.x, v0.y, v1);
}

__host__ __device__ inline float2 operator*(const float2& a, const float2& b)
{
	return make_float2(a.x * b.x, a.y * b.y);
}

__host__ __device__ inline float2 operator*(const float2& a, const float s)
{
	return make_float2(a.x * s, a.y * s);
}

__host__ __device__ inline float2 operator*(const float s, const float2& a)
{
	return make_float2(a.x * s, a.y * s);
}

__host__ __device__ inline void operator*=(float2& a, const float2& s)
{
	a.x *= s.x;
	a.y *= s.y;
}

__host__ __device__ inline void operator*=(float2& a, const float s)
{
	a.x *= s;
	a.y *= s;
}

/** divide 
* @{
*/
__host__ __device__ inline float2 operator/(const float2& a, const float2& b)
{
	return make_float2(a.x / b.x, a.y / b.y);
}

__host__ __device__ inline float2 operator/(const float2& a, const float s)
{
	float inv = 1.0f / s;
	return a * inv;
}

__host__ __device__ inline float2 operator/(const float s, const float2& a)
{
	return make_float2(s / a.x, s / a.y);
}

__host__ __device__ inline void operator/=(float2& a, const float s)
{
	float inv = 1.0f / s;
	a *= inv;
}

/** add 
* @{
*/
__host__ __device__ inline float3 operator+(const float3& a, const float3& b)
{
	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ inline float3 operator+(const float3& a, const float b)
{
	return make_float3(a.x + b, a.y + b, a.z + b);
}

__host__ __device__ inline float3 operator+(const float a, const float3& b)
{
	return make_float3(a + b.x, a + b.y, a + b.z);
}

__host__ __device__ inline void operator+=(float3& a, const float3& b)
{
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}
/** @} */

/** subtract 
* @{
*/
__host__ __device__ inline float3 operator-(const float3& a, const float3& b)
{
	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ inline float3 operator-(const float3& a, const float b)
{
	return make_float3(a.x - b, a.y - b, a.z - b);
}

__host__ __device__ inline float3 operator-(const float a, const float3& b)
{
	return make_float3(a - b.x, a - b.y, a - b.z);
}

__host__ __device__ inline void operator-=(float3& a, const float3& b)
{
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
}
/** @} */

/** multiply 
* @{
*/
__host__ __device__ inline float3 operator*(const float3& a, const float3& b)
{
	return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__host__ __device__ inline float3 operator*(const float3& a, const float s)
{
	return make_float3(a.x * s, a.y * s, a.z * s);
}

__host__ __device__ inline float3 operator*(const float s, const float3& a)
{
	return make_float3(a.x * s, a.y * s, a.z * s);
}

__host__ __device__ inline void operator*=(float3& a, const float3& s)
{
	a.x *= s.x;
	a.y *= s.y;
	a.z *= s.z;
}

__host__ __device__ inline void operator*=(float3& a, const float s)
{
	a.x *= s;
	a.y *= s;
	a.z *= s;
}
/** @} */

/** divide 
* @{
*/
__host__ __device__ inline float3 operator/(const float3& a, const float3& b)
{
	return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}

__host__ __device__ inline float3 operator/(const float3& a, const float s)
{
	float inv = 1.0f / s;
	return a * inv;
}

__host__ __device__ inline float3 operator/(const float s, const float3& a)
{
	return make_float3(s / a.x, s / a.y, s / a.z);
}

__host__ __device__ inline void operator/=(float3& a, const float s)
{
	float inv = 1.0f / s;
	a *= inv;
}

//vector operations
__host__ __device__ float dot(const float3& a, const float3& b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__host__ __device__ float3 normalize(const float3& v)
{
	float invLen = 1.0f / sqrtf(dot(v, v));
	return v * invLen;
}

//color helpers
__forceinline__ __device__ float3 toSRGB(const float3& c)
{
	float invGamma = 1.0f / 2.4f;
	float3 powed = make_float3(powf(c.x, invGamma), powf(c.y, invGamma), powf(c.z, invGamma));
	return make_float3(c.x < 0.0031308f ? 12.92f * c.x : 1.055f * powed.x - 0.055f,
					   c.y < 0.0031308f ? 12.92f * c.y : 1.055f * powed.y - 0.055f,
					   c.z < 0.0031308f ? 12.92f * c.z : 1.055f * powed.z - 0.055f);
}

__forceinline__ __device__ float clamp(const float f, const float a, const float b)
{
	return fmaxf(a, fminf(f, b));
}

__forceinline__ __device__ float3 clamp(const float3& v, const float a, const float b)
{
	return make_float3(clamp(v.x, a, b), clamp(v.y, a, b), clamp(v.z, a, b));
}

__forceinline__ __device__ unsigned char quantizeUnsigned8Bits(float x)
{
	x = clamp(x, 0.0f, 1.0f);
	enum
	{
		N = (1 << 8) - 1,
		Np1 = (1 << 8)
	};
	return (unsigned char)min((unsigned int)(x * (float)Np1), (unsigned int)N);
}

__forceinline__ __device__ uchar4 make_color(const float3& c)
{
	// first apply gamma, then convert to unsigned char
	float3 srgb = toSRGB(clamp(c, 0.0f, 1.0f));
	return make_uchar4(quantizeUnsigned8Bits(srgb.x),
					   quantizeUnsigned8Bits(srgb.y),
					   quantizeUnsigned8Bits(srgb.z),
					   255u);
}

__forceinline__ __device__ uchar4 make_color(const float4& c)
{
	return make_color(make_float3(c.x, c.y, c.z));
}

static __forceinline__ __device__ void
computeRay(uint3 idx, uint3 dim, float3& origin, float3& direction)
{
	const float3 U = params.cam_u;
	const float3 V = params.cam_v;
	const float3 W = params.cam_w;
	const float2 d = make_float2(static_cast<float>(idx.x) / static_cast<float>(dim.x),
								 static_cast<float>(idx.y) / static_cast<float>(dim.y));

	origin = params.cam_eye;
	direction = normalize(d.x * U + d.y * V + W);
}

extern "C" __global__ void __raygen__rg()
{
	// Lookup our location within the launch grid
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	// Map our launch idx to a screen location and create a ray from the camera
	// location through the screen
	float3 ray_origin, ray_direction;
	computeRay(idx, dim, ray_origin, ray_direction);

	// Trace the ray against our scene hierarchy
	unsigned int p0, p1, p2;
	optixTrace(params.handle,
			   ray_origin,
			   ray_direction,
			   0.0f, // Min intersection distance
			   1e16f, // Max intersection distance
			   0.0f, // rayTime -- used for motion blur
			   OptixVisibilityMask(255), // Specify always visible
			   OPTIX_RAY_FLAG_NONE,
			   0, // SBT offset   -- See SBT discussion
			   1, // SBT stride   -- See SBT discussion
			   0, // missSBTIndex -- See SBT discussion
			   p0,
			   p1,
			   p2);
	float3 result;
	result.x = __uint_as_float(p0);
	result.y = __uint_as_float(p1);
	result.z = __uint_as_float(p2);

	// Record results in our output raster
	params.image[idx.y * params.image_width + idx.x] = make_color(result);
}

extern "C" __global__ void __miss__ms()
{
	MissData* miss_data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
	setPayload(miss_data->bg_color);
}

extern "C" __global__ void __closesthit__ch()
{
	// When built-in triangle intersection is used, a number of fundamental
	// attributes are provided by the OptiX API, indlucing barycentric coordinates.
	const float2 barycentrics = optixGetTriangleBarycentrics();

	setPayload(make_float3(barycentrics, 1.0f));
}
